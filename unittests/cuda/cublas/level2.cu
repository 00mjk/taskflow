#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cublasflow.hpp>

// ----------------------------------------------------------------------------
// Testcase: gemv and c_gemv
// ----------------------------------------------------------------------------

template <typename T>
void gemv(
  bool row_major,
  const int M, 
  const int N, 
  const std::vector<T>& hA,
  const std::vector<T>& hx,
  const std::vector<T>& golden,
  bool trans
) {

  for(size_t d=0; d<tf::cuda_get_num_devices(); d++) {
  tf::Taskflow taskflow;
  tf::Executor executor;

    auto dA = tf::cuda_malloc_device<T>(M*N, d);
    auto dAlpha = tf::cuda_malloc_device<T>(1, d);
    auto dBeta  = tf::cuda_malloc_device<T>(1, d);

    T* hy;
    T* dx;
    T* dy;

    if(trans) {
      hy = new T[N];
      dx = tf::cuda_malloc_device<T>(M, d);
      dy = tf::cuda_malloc_device<T>(N, d);
    }
    else {
      hy = new T[M];
      dx = tf::cuda_malloc_device<T>(N, d);
      dy = tf::cuda_malloc_device<T>(M, d);
    }

    auto cudaflow = taskflow.emplace_on([=](tf::cudaFlow& cf){

      REQUIRE(tf::cuda_get_device() == d);
      
      auto copyA = cf.copy(dA, hA.data(), M*N);

      tf::cudaTask copyx;

      (trans) ? copyx = cf.copy(dx, hx.data(), M)
              : copyx = cf.copy(dx, hx.data(), N);

      auto alpha = cf.single_task([=] __device__ () { *dAlpha = 1; });
      auto beta  = cf.single_task([=] __device__ () { *dBeta  = 0; });

      tf::cudaTask gemv; 
      
      if(trans) {        
        if(row_major) {       // C = A^T * x (r-major)
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->c_gemv(
              HIPBLAS_OP_T,
              M, N, dAlpha, dA, N, dx, 1, dBeta, dy, 1 
            );
          });
        }
        else {
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->gemv(
              HIPBLAS_OP_N,
              N, M, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
      }
      else {            
        if(row_major) {       // C = A * x (r-major)
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->c_gemv(
              HIPBLAS_OP_N,
              M, N, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
        else {
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->gemv(
              HIPBLAS_OP_T,
              N, M, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
      }
      
      tf::cudaTask copyy; 
      (trans) ? copyy = cf.copy(hy, dy, N)
              : copyy = cf.copy(hy, dy, M);

      gemv.precede(copyy)
          .succeed(copyA, copyx, alpha, beta);
    }, d);

    auto verify = taskflow.emplace([=, &golden](){
      for(size_t i=0; i<golden.size(); i++) {
        //std::cerr << hy[i] << " ";
        REQUIRE(std::fabs(hy[i]-golden[i]) < 0.0001);
      }
      //std::cerr << '\n';
      tf::cuda_free(dA);
      tf::cuda_free(dx);
      tf::cuda_free(dy);
      tf::cuda_free(dAlpha);
      tf::cuda_free(dBeta);
      delete [] hy;
    });
    
    cudaflow.precede(verify);

  executor.run(taskflow).wait();
  }

}

template <typename T>
void gemv_test(bool row_major, bool trans) {

  int M = 3, N = 4;

  const std::vector<T> hA = {
    11, 12, 13, 14,
    15, 16, 17, 18,
    19, 20, 21, 22
  };  // 3x4

  std::vector<T> hx;
  std::vector<T> golden;

  //ha.T * hx
  if(trans) {
    hx = {11, 12, 13};
    golden = {548, 584, 620, 656};
  }
  else {
    hx = {11, 12, 13, 14};
    golden = {630, 830, 1030};
  }

  gemv<T>(row_major, M, N, hA, hx, golden, trans);
}

// gemv (column-major)
TEST_CASE("gemv_n.float") {
  gemv_test<float>(false, false);
}

TEST_CASE("gemv_n.double") {
  gemv_test<double>(false, false);
}

TEST_CASE("gemv_t.float") {
  gemv_test<float>(false, true);
}

TEST_CASE("gemv_t.double") {
  gemv_test<double>(false, true);
}

// gemv (row-major)
TEST_CASE("c_gemv_n.float") {
  gemv_test<float>(true, false);
}

TEST_CASE("c_gemv_n.double") {
  gemv_test<double>(true, false);
}

TEST_CASE("c_gemv_t.float") {
  gemv_test<float>(true, true);
}

TEST_CASE("c_gemv_t.double") {
  gemv_test<double>(true, true);
}

// ----------------------------------------------------------------------------
// trsv
// ----------------------------------------------------------------------------

template <typename T>
void c_trsv_test() {

  int N = 3;

  const std::vector<T> hA = {
    2, 0, 0,
    1, 2, 0,
    1, 1, 2
  };

  const std::vector<T> hB = {
    5,
    4,
    7
  };

  const std::vector<T> sol = {
    2.5,
    0.75,
    1.875
  };

  std::vector<T> res(N, 0);

  tf::Taskflow taskflow;
  tf::Executor executor;

  auto dA = tf::cuda_malloc_device<T>(hA.size());
  auto dB = tf::cuda_malloc_device<T>(hB.size());

  taskflow.emplace([&](tf::cudaFlowCapturer& capturer){
    auto blas = capturer.make_capturer<tf::cublasFlowCapturer>();
    auto h2dA = blas->copy(dA, hA.data(), hA.size());
    auto h2dB = blas->copy(dB, hB.data(), hB.size());
    auto trsv = blas->c_trsv(HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 
      N, dA, N, dB, 1
    );
    auto d2h = blas->copy(res.data(), dB, res.size());

    trsv.succeed(h2dA, h2dB)
        .precede(d2h);
  });

  executor.run(taskflow).wait();
  
  for(size_t i=0; i<res.size(); ++i) {
    //std::cout << res[i] << '\n';
    REQUIRE(std::fabs(res[i] - sol[i]) < 0.0001);
  }

}

TEST_CASE("c_trsv.float") {
  c_trsv_test<float>();
}

TEST_CASE("c_trsv.double") {
  c_trsv_test<double>();
}





