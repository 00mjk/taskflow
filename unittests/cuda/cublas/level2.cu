#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cublas.hpp>

// ----------------------------------------------------------------------------
// Testcase: gemv and c_gemv
// ----------------------------------------------------------------------------

template <typename T>
void gemv(
  bool row_major,
  const int M, 
  const int N, 
  const std::vector<T>& hA,
  const std::vector<T>& hx,
  const std::vector<T>& golden,
  bool trans
) {

  for(size_t d=0; d<tf::cuda_get_num_devices(); d++) {
  tf::Taskflow taskflow;
  tf::Executor executor;

    auto dA = tf::cuda_malloc_device<T>(M*N, d);
    auto dAlpha = tf::cuda_malloc_device<T>(1, d);
    auto dBeta  = tf::cuda_malloc_device<T>(1, d);

    T* hy;
    T* dx;
    T* dy;

    if(trans) {
      hy = new T[N];
      dx = tf::cuda_malloc_device<T>(M, d);
      dy = tf::cuda_malloc_device<T>(N, d);
    }
    else {
      hy = new T[M];
      dx = tf::cuda_malloc_device<T>(N, d);
      dy = tf::cuda_malloc_device<T>(M, d);
    }

    auto cudaflow = taskflow.emplace_on([=](tf::cudaFlow& cf){

      REQUIRE(tf::cuda_get_device() == d);
      
      auto copyA = cf.copy(dA, hA.data(), M*N);

      tf::cudaTask copyx;

      (trans) ? copyx = cf.copy(dx, hx.data(), M)
              : copyx = cf.copy(dx, hx.data(), N);

      auto alpha = cf.single_task([=] __device__ () { *dAlpha = 1; });
      auto beta  = cf.single_task([=] __device__ () { *dBeta  = 0; });

      tf::cudaTask gemv; 
      
      if(trans) {        
        if(row_major) {       // C = A^T * x (r-major)
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->c_gemv(
              HIPBLAS_OP_T,
              M, N, dAlpha, dA, N, dx, 1, dBeta, dy, 1 
            );
          });
        }
        else {
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->gemv(
              HIPBLAS_OP_N,
              N, M, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
      }
      else {            
        if(row_major) {       // C = A * x (r-major)
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->c_gemv(
              HIPBLAS_OP_N,
              M, N, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
        else {
          gemv = cf.capture([&](tf::cudaFlowCapturer& cap){
            cap.make_capturer<tf::cublasFlowCapturer>()->gemv(
              HIPBLAS_OP_T,
              N, M, dAlpha, dA, N, dx, 1, dBeta, dy, 1
            );
          });
        }
      }
      
      tf::cudaTask copyy; 
      (trans) ? copyy = cf.copy(hy, dy, N)
              : copyy = cf.copy(hy, dy, M);

      gemv.precede(copyy)
          .succeed(copyA, copyx, alpha, beta);
    }, d);

    auto verify = taskflow.emplace([=, &golden](){
      for(size_t i=0; i<golden.size(); i++) {
        //std::cerr << hy[i] << " ";
        REQUIRE(std::fabs(hy[i]-golden[i]) < 0.0001);
      }
      //std::cerr << '\n';
      tf::cuda_free(dA);
      tf::cuda_free(dx);
      tf::cuda_free(dy);
      tf::cuda_free(dAlpha);
      tf::cuda_free(dBeta);
      delete [] hy;
    });
    
    cudaflow.precede(verify);

  executor.run(taskflow).wait();
  }

}

template <typename T>
void gemv_test(bool row_major, bool trans) {

  int M = 3, N = 4;

  const std::vector<T> hA = {
    11, 12, 13, 14,
    15, 16, 17, 18,
    19, 20, 21, 22
  };  // 3x4

  std::vector<T> hx;
  std::vector<T> golden;

  //ha.T * hx
  if(trans) {
    hx = {11, 12, 13};
    golden = {548, 584, 620, 656};
  }
  else {
    hx = {11, 12, 13, 14};
    golden = {630, 830, 1030};
  }

  gemv<T>(row_major, M, N, hA, hx, golden, trans);
}

// gemv (column-major)
TEST_CASE("gemv_n.float") {
  gemv_test<float>(false, false);
}

TEST_CASE("gemv_n.double") {
  gemv_test<double>(false, false);
}

TEST_CASE("gemv_t.float") {
  gemv_test<float>(false, true);
}

TEST_CASE("gemv_t.double") {
  gemv_test<double>(false, true);
}

// gemv (row-major)
TEST_CASE("c_gemv_n.float") {
  gemv_test<float>(true, false);
}

TEST_CASE("c_gemv_n.double") {
  gemv_test<double>(true, false);
}

TEST_CASE("c_gemv_t.float") {
  gemv_test<float>(true, true);
}

TEST_CASE("c_gemv_t.double") {
  gemv_test<double>(true, true);
}
